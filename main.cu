#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include "H5Cpp.h"
#include <vector>
#include <random>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <omp.h>

using namespace std;

class stopwatch
{
  struct timespec start_time, stop_time;
  double elapsed;
  bool running = false;
  bool onscreen = false;
public:
  void click()
  {
    if(running)
    {
      clock_gettime(CLOCK_MONOTONIC, &(this->stop_time));
      this->running = false;
      this->onscreen = true;
    }
    else
    {
      clock_gettime(CLOCK_MONOTONIC, &(this->start_time));
      this->running = true;
      this->onscreen = false;
    }
  }
  double check()
  {
    if(onscreen)
    {
      this->elapsed = (this->stop_time.tv_sec - this->start_time.tv_sec);
      this->elapsed += (this->stop_time.tv_nsec - this->start_time.tv_nsec) / 1000000000.0;
      return this->elapsed*1000;
    }
    else
      return 0.0;
  }
  void print_time()
  {
    std::cout<<this->check()<<"ms\n";
  }
};

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
	float* fp = new float[6]; 
	memcpy(fp,ppm,6*sizeof(float)); 
	for(int i = 0; i < 6; i++)
	{
		fp[i] *= (FIELD_STRENGTH*42.577);
	}
	return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};



float* signal_equation(const float TR, const float alpha, const float TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi){	
	float sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	float sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	
	float* sig = new float[2];
	sig[0] = sR;
	sig[1] = sI;
	return sig;
}

float* noiseless_signal_vector(const int N, const float* TR, const float* alpha, const float* TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi, const int NUMTHREADS)
{
	float* vect_out = new float[2*N]{};

	// for(int n=0; n<N; n++)
	// {
	// 	float* sig = signal_equation(TR[n], alpha[n], TE[n], beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi);
	// 	vect_out[n] = sig[0];
	// 	vect_out[n+N] = sig[1];
	// }
	// return vect_out;

	float* sig;

	#pragma omp parallel shared(vect_out, TR, alpha, TE, beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi) private(sig) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for(int n=0; n<N; n++)
		{
			sig = signal_equation(TR[n], alpha[n], TE[n], beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi);
			vect_out[n] = sig[0];
			vect_out[n+N] = sig[1];
		}

	}
	return vect_out;
}



void pvect(vector<float> v)
{
	for (auto entry:v)
	{
		cout<<entry<<" ";
	}
	cout<<endl;
}

void parray(float* v, int N)
{
	for (int i =0; i<N; i++)
	{
		cout<<v[i]<<" ";
	}
	cout<<endl;
}

float* noise_vector(const int N, const float stddev, const int NUMTHREADS)
{  
	float* vect = new float[N]{}; 

	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	default_random_engine generator (seed);
	normal_distribution<float> distribution(0.0,stddev);


	#pragma omp parallel shared(vect, stddev) private(seed) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for (int n = 0; n < N; n++)
		{
			vect[n]=distribution(generator);
		}

	}


	return vect;
}


int main(int argc, char const *argv[])
{
	int NUMTHREADS;
	if (argc<2)
	{
		cout<<"Usage ./main N (N=number of threads)";
		return 1;
	}
	else
	{
		NUMTHREADS = atoi(argv[1]);

	}

	int NPs = 3; // Number of TR/FlipAngle Pairs
	int NTEs = 6;
	int NACQS = NPs*NTEs;

	float tes[NTEs] = {1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3};
	float trs[NPs] = {5e-3,10e-3,15e-3};
	float tips[NPs] = {6*PI/180,12*PI/180,80*PI/180};

	float TES[NACQS]{}; for (int i = 0; i < NACQS; i++) { TES[i] = tes[i%NTEs];}
	float TRS[NACQS]{}; for (int i = 0; i < NACQS; i++) { TRS[i] = trs[i/NTEs];}
	float TIPS[NACQS]{}; for (int i = 0; i < NACQS; i++) { TIPS[i] = tips[i/NTEs];}


	struct stopwatch sw;
	sw.click();
	float* pure_signal = noiseless_signal_vector(NACQS, TRS, TIPS, TES,1.0,312e-3,822e-3,50,9050,30,0,0, NUMTHREADS);

	float simulated_signal[2*NACQS]{};
	thrust::transform(thrust::host, pure_signal, pure_signal+(2*NACQS), noise_vector((2*NACQS),1.0, NUMTHREADS), simulated_signal, thrust::plus<float>());
	sw.click();
	sw.print_time();
	parray(simulated_signal,(2*NACQS));

	return 0;
}




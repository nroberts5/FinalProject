#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include "H5Cpp.h"
#include <vector>
#include <random>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

using namespace std;

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
	float* fp = new float[6]; 
	memcpy(fp,ppm,6*sizeof(float)); 
	for(int i = 0; i < 6; i++)
	{
		fp[i] *= (FIELD_STRENGTH*42.577);
	}
	return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};



float* signal_equation(const float TR, const float alpha, const float TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi){	
	float sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	float sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	
	float* sig = new float[2];
	sig[0] = sR;
	sig[1] = sI;
	return sig;
}

float* noiseless_signal_vector(const int N, const float* TR, const float* alpha, const float* TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi)
{
	float* vect_out = new float[2*N]{};

	for(int n=0; n<N; n++)
	{
		float* sig = signal_equation(TR[n], alpha[n], TE[n], beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi);
		vect_out[n] = sig[0];
		vect_out[n+N] = sig[1];
		// cout<<sig[0]<<"+j"<<sig[1]<<endl;
	}
	return vect_out;
}



void pvect(vector<float> v)
{
	for (auto entry:v)
	{
		cout<<entry<<" ";
	}
	cout<<endl;
}

void parray(float* v, int N)
{
	for (int i =0; i<N; i++)
	{
		cout<<v[i]<<" ";
	}
	cout<<endl;
}

float* noise_vector(const int N, const float stddev)
{  
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  	default_random_engine generator (seed);
	normal_distribution<float> distribution(0.0,stddev);
	float* vect = new float[N]{}; for(int n=0;n<N;n++){vect[n]=distribution(generator);}
	return vect;
}


int main(int argc, char const *argv[])
{
	int Nacqs = 3;
	int NTEs = 6;
	int N = Nacqs*NTEs;

	float tes[NTEs] = {1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3};
	float trs[Nacqs] = {5e-3,10e-3,15e-3};
	float tips[Nacqs] = {6*PI/180,12*PI/180,80*PI/180};

	float TES[N]{}; for (int i = 0; i < N; i++) { TES[i] = tes[i%NTEs];}
	float TRS[N]{}; for (int i = 0; i < N; i++) { TRS[i] = trs[i/NTEs];}
	float TIPS[N]{}; for (int i = 0; i < N; i++) { TIPS[i] = tips[i/NTEs];}

	float* pure_signal = noiseless_signal_vector(N, TRS, TIPS, TES,1.0,312e-3,822e-3,50,9050,30,0,0);

	float simulated_signal[2*N]{};
	thrust::transform(thrust::host, pure_signal, pure_signal+(2*N), noise_vector((2*N),1.0), simulated_signal, thrust::plus<float>());
	parray(simulated_signal,(2*N));

	return 0;
}




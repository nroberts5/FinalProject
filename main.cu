#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include "H5Cpp.h"
#include <vector>
#include <random>
#include <chrono>
#include <thrust/transform.h>
#include <omp.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <cstdlib>


using namespace std;

class stopwatch
{
  struct timespec start_time, stop_time;
  double elapsed;
  bool running = false;
  bool onscreen = false;
public:
  void click()
  {
    if(running)
    {
      clock_gettime(CLOCK_MONOTONIC, &(this->stop_time));
      this->running = false;
      this->onscreen = true;
    }
    else
    {
      clock_gettime(CLOCK_MONOTONIC, &(this->start_time));
      this->running = true;
      this->onscreen = false;
    }
  }
  double check()
  {
    if(onscreen)
    {
      this->elapsed = (this->stop_time.tv_sec - this->start_time.tv_sec);
      this->elapsed += (this->stop_time.tv_nsec - this->start_time.tv_nsec) / 1000000000.0;
      return this->elapsed*1000;
    }
    else
      return 0.0;
  }
  void print_time()
  {
    std::cout<<this->check()<<"ms\n";
  }
};

int rc2ii(int row, int col, int N)
{
    return row*N + col;
}

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
	float* fp = new float[6]; 
	memcpy(fp,ppm,6*sizeof(float)); 
	for(int i = 0; i < 6; i++)
	{
		fp[i] *= (FIELD_STRENGTH*42.577);
	}
	return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};



float* signal_equation(const float TR, const float alpha, const float TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi){	
	float sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	float sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
	
	float* sig = new float[2];
	sig[0] = sR;
	sig[1] = sI;
	return sig;
}

float* noiseless_signal_vector(const int N, const float* TR, const float* alpha, const float* TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi, const int NUMTHREADS)
{
	float* vect_out = new float[2*N]{};
	float* sig;

	#pragma omp parallel shared(vect_out) private(sig) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for(int n=0; n<N; n++)
		{
			sig = signal_equation(TR[n], alpha[n], TE[n], beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi);
			vect_out[n] = sig[0];
			vect_out[n+N] = sig[1];
		}

	}
	return vect_out;
}



void pvect(vector<float> v)
{
	for (auto entry:v)
	{
		cout<<entry<<" ";
	}
	cout<<endl;
}

void parray(float* v, int N)
{
	for (int i =0; i<N; i++)
	{
		cout<<v[i]<<" ";
	}
	cout<<endl;
}

void printmat(float* v, int rows, int cols)
{
	for (int row = 0; row < rows; row++)
	{
		for (int col = 0; col < cols; col++)
		{
			cout<<v[col+row*cols]<<" ";
		}
		cout<<"\n";
	}
}

unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();

default_random_engine generator (seed);


float* noise_vector(const int N, const float stddev, const int NUMTHREADS)
{  
	float* vect = new float[N]{}; 

	normal_distribution<float> distribution(0.0,1.0);


	#pragma omp parallel shared(vect, stddev) private(seed) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for (int n = 0; n < N; n++)
		{
			vect[n]=distribution(generator);
		}

	}


	return vect;
}

float* add_noise_2_vector(const float* vect_in, const int N, const float stddev, const int NUMTHREADS)
{  
	float* vect_out = new float[N]{}; 
	normal_distribution<float> distribution(0.0,1.0);
	#pragma omp parallel shared(vect, stddev) private(seed) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for (int n = 0; n < N; n++)
		{
			vect_out[n]=vect_in[n]+distribution(generator);
		}

	}
	return vect_out;
}

// thrust::minstd_rand rng (seed);
// thrust::random::normal_distribution<float> dist(0.0, 1.0);
// struct GenRand
// {
// 	__device__ __host__ 
// 	float operator ()()
// 	{
// 		return dist(rng);
// 	}
// };


float* tile_vector(const float* vect, const int n, const int numREPS, const int NUMTHREADS)
{
	float* vect_out = new float[numREPS*n];
	
	#pragma omp parallel shared(vect_out, numREPS, n) num_threads(NUMTHREADS)
	{
		#pragma omp for nowait
		for (int i = 0; i < numREPS; i++)
		{
			memcpy(&vect_out[i*n], vect, n*sizeof(*vect));
		}
	}

	return vect_out;
}


int main(int argc, char const *argv[])
{
	int NUMTHREADS;
	int NSIMS;

	if (argc<3)
	{
		cout<<"Usage ./main N (N=number of threads)\n";
		return 1;
	}
	else
	{
		NUMTHREADS = atoi(argv[1]);
		NSIMS = atoi(argv[2]);
	}

	int NPs = 3; // Number of TR/FlipAngle Pairs
	int NTEs = 6; // Number of Echoes
	int NACQS = NPs*NTEs; //

	float tes[NTEs] = {1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3};
	float trs[NPs] = {5e-3,10e-3,15e-3};
	float tips[NPs] = {6*PI/180,12*PI/180,80*PI/180};

	float TES[NACQS]{}; for (int i = 0; i < NACQS; i++) { TES[i] = tes[i%NTEs];}
	float TRS[NACQS]{}; for (int i = 0; i < NACQS; i++) { TRS[i] = trs[i/NTEs];}
	float TIPS[NACQS]{}; for (int i = 0; i < NACQS; i++) { TIPS[i] = tips[i/NTEs];}


	struct stopwatch sw;
	sw.click();
	float* pure_signal = noiseless_signal_vector(NACQS, TRS, TIPS, TES,1.0,312e-3,822e-3,50,9050,30,0,0, NUMTHREADS);
	sw.click();
	cout<<"Making orignal signal: "<<sw.check()<<endl;

	sw.click();
	float* pure_signal_mat = tile_vector(pure_signal,2*NACQS, NSIMS, NUMTHREADS);
	sw.click();
	cout<<"Repeating the Signal: "<<sw.check()<<endl;

	// sw.click();
	// float* nv = noise_vector((2*NACQS*NSIMS), 1.0, NUMTHREADS);
	// sw.click();
	// cout<<"Creating Noise Vector: "<<sw.check()<<endl;

	sw.click();
	float* simsig = add_noise_2_vector(pure_signal_mat, (2*NACQS*NSIMS), 1.0, NUMTHREADS);
	sw.click();
	cout<<"Creating Simulated Signal with Noise Vector: "<<sw.check()<<endl;

	// sw.click();
	// float rd[2*NACQS*NSIMS];
	// thrust::generate(thrust::host, rd, rd+(2*NACQS*NSIMS), GenRand());
	// sw.click();
	// cout<<"Thrust Noise Vector: "<<sw.check()<<endl;

	// sw.click();
	// float simulated_signal[2*NACQS*NSIMS]{};
	// cout<<"pure_signal_mat size: "<<sizeof(pure_signal_mat)/sizeof(*pure_signal_mat)<<endl;
	// cout<<"nv size: "<<sizeof(nv)/sizeof(*nv)<<endl;
	// cout<<"simulated_signal size: "<<sizeof(simulated_signal)/sizeof(*simulated_signal)<<endl;

	// for (int i = 0; i < 2*NACQS*NSIMS; i++)
	// {
	// 	simulated_signal[i] = pure_signal_mat[i]+nv[i];
	// }

	// thrust::host_vector<float> h_pure_sig(pure_signal_mat,pure_signal_mat+(2*NACQS*NSIMS));
	// thrust::host_vector<float> h_sim_sig(simulated_signal,simulated_signal+(2*NACQS*NSIMS));
	// thrust::host_vector<float> h_noise(nv,nv+(2*NACQS*NSIMS));
	// thrust::transform(thrust::host, pure_signal_mat, pure_signal_mat+(2*NACQS*NSIMS), nv, simulated_signal, thrust::plus<float>());
	// sw.click();
	
	// cout<<"Adding the Noise: "<<sw.check()<<endl<<endl;


	return 0;
}




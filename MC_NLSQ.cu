#include "hip/hip_runtime.h"
#include "./dlib-19.7/dlib/optimization.h"
#include <iostream>
#include <vector>
#include <omp.h>
#include <chrono>
#include <random>
#include "tools.hpp"
#include <fstream>
#include <string>


using namespace std;
using namespace dlib;

// ----------------------------------------------------------------------------------------

typedef matrix<double,4,1> input_vector;
typedef matrix<double,8,1> parameter_vector;
typedef matrix<double,0,1> column_vector;

// ----------------------------------------------------------------------------------------

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
#define STD_DEV 1.0

unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
default_random_engine generator (seed);


float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
    float* fp = new float[6]; 
    memcpy(fp,ppm,6*sizeof(float)); 
    for(int i = 0; i < 6; i++)
    {
        fp[i] *= (FIELD_STRENGTH*42.577);
    }
    return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};

// ----------------------------------------------------------------------------------------

double signal (const column_vector& acq_params, const column_vector& unknowns)
{
    const double beta = unknowns(0);
    const double T1__F = unknowns(1);
    const double T1__W = unknowns(2);
    const double rho__F = unknowns(3);
    const double rho__W = unknowns(4);
    const double R2s = unknowns(5);
    const double phi = unknowns(6);
    const double psi = unknowns(7);

    const double TR = acq_params(0);
    const double alpha = acq_params(1);
    const double TE = acq_params(2);
    const double real_part = acq_params(3);

    // compute signal model function and return the result
    double sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    double sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    
    return (real_part)*sR+(1-real_part)*sI;
}

double residual (const std::pair<column_vector, double>& data, const column_vector& params)
{
    return signal(data.first, params) - data.second;
}

std::vector<pair<input_vector,double>> signal_pairs(const int NACQS, const int NTES, const column_vector& trs, const column_vector& tips, const column_vector& tes, const column_vector& unknowns)
{
    std::vector<pair<input_vector,double>> data_samples;
    input_vector input;
    for (int nacq = 0; nacq < NACQS; nacq++)
    {
        for (int nte = 0; nte < NTES; nte++)
        {
            for (int real_part = 1; real_part >=0; real_part--)
            {
                input = trs(nacq), tips(nacq), tes(nte), real_part;
                const double sim_sig = signal(input,unknowns);
                data_samples.push_back(make_pair(input,sim_sig));
            }
        }
    }
    return data_samples;
}

std::vector<pair<input_vector,double>> add_noise2signal_pairs(const std::vector<pair<input_vector,double>> data_samples, const double stddev)
{
    normal_distribution<double> distribution(0.0,stddev);
    std::vector<pair<input_vector,double>> noisy_data_samples;
    for (auto ind_pair : data_samples)
    {
        const input_vector input = ind_pair.first;
        double sim_sig = ind_pair.second;
        noisy_data_samples.push_back(make_pair(input,sim_sig+distribution(generator)));
    }
    return noisy_data_samples;
}


column_vector NLSQ(std::vector<pair<input_vector,double>> noiseless_data_samples, const double std_dev_noise, const parameter_vector& unknowns)
{
    std::vector<pair<input_vector,double>> noisy_data_samples = add_noise2signal_pairs(noiseless_data_samples,std_dev_noise);
    parameter_vector x;
    x=1, 300e-3, 800e-3, 5000, 5000, 0, 0, 0;
    solve_least_squares_lm(objective_delta_stop_strategy(1e-10)/*.be_verbose()*/, 
                           residual,
                           derivative(residual),
                           noisy_data_samples,
                           x);
    return x;
}

int main(int argc, char const *argv[])
{
    int NUMTHREADS;
    int NSIMS;
    int NUMTRIALS = 1;

    if (argc<3)
    {
        cout<<"Usage ./MC_NLSQ NUMTHREADS NSIMS\n";
        return 1;
    }
    else
    {
        NUMTHREADS = atoi(argv[1]);
        NSIMS = atoi(argv[2]);
        // NUMTRIALS = atoi(argv[3]);
    }

    int NACQS = 3; // Number of TR/FlipAngle Pairs
    int NTES = 6; // Number of Echoes


    // Define Acquisition Parameters
    column_vector trs(NACQS); trs = 5e-3,10e-3,15e-3;
    column_vector tips(NACQS); tips = 6*PI/180,12*PI/180,80*PI/180;
    column_vector tes(NTES); tes = 1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3;

    // Define Unknown Parameters (set their true values for simulation)
    double beta=1.0, T1__F=312e-3, T1__W=822e-3, rho__F=50, rho__W=9050, R2s=30, phi=0, psi=0;
    parameter_vector unknowns; unknowns = beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi;

    // Generate the noiseless expected signal values in a structure usable by the NLSQ function
    std::vector<pair<input_vector,double>> noiseless_data_samples = signal_pairs(NACQS, NTES, trs, tips, tes, unknowns);
    
    struct stopwatch sw;
    double param_estimates[8][NSIMS];

    // Outler loop is for timing analysis
    std::vector<double> timings;
    for (int trial = 0; trial < NUMTRIALS; trial++)
    {
        sw.click();
        // OpenMP Parallel Region
        #pragma omp parallel num_threads(NUMTHREADS)
        {
            #pragma omp for nowait
            for (int i = 0; i < NSIMS; i++)
            {
                column_vector x = NLSQ(noiseless_data_samples, 1.0, unknowns);

                param_estimates[0][i] = x(0);
                param_estimates[1][i] = x(1);
                param_estimates[2][i] = x(2);
                param_estimates[3][i] = x(3);
                param_estimates[4][i] = x(4);
                param_estimates[5][i] = x(5);
                param_estimates[6][i] = x(6);
                param_estimates[7][i] = x(7);
            }
        }
        sw.click();
        timings.push_back(sw.check());

        string names[] = {"beta.out", "T1__F.out", "T1__W.out", "rho__F.out", "rho__W.out", "R2s.out", "phi.out", "psi.out"};
        
        for (int n = 0; n < 8; n++)
        {
            ofstream outfile;
            outfile.open (names[n]);
            for (int i = 0; i < NSIMS; i++)
            {
                outfile<<param_estimates[n][i]<<"\n";
            }
            outfile.close();
        }
    }

    column_vector t = mat(timings);
    double minTIME = dlib::min(t);
    cout<<NSIMS<<", "<<NUMTHREADS<<", "<<minTIME/1000.0<<"s"<<", <"<<NUMTRIALS<<" trial(s)>"<<endl;

    return 0;
}

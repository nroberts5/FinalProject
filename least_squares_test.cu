#include "hip/hip_runtime.h"
// The contents of this file are in the public domain. See LICENSE_FOR_EXAMPLE_PROGRAMS.txt
/*

    This is an example illustrating the use the general purpose non-linear 
    least squares optimization routines from the dlib C++ Library.

    This example program will demonstrate how these routines can be used for data fitting.
    In particular, we will generate a set of data and then use the least squares  
    routines to infer the parameters of the model which generated the data.
*/


#include "./dlib-19.7/dlib/optimization.h"
#include <iostream>
#include <vector>
#include <omp.h>
#include <chrono>
#include <random>
#include "tools.hpp"


using namespace std;
using namespace dlib;

// ----------------------------------------------------------------------------------------

typedef matrix<double,4,1> input_vector;
typedef matrix<double,8,1> parameter_vector;
typedef matrix<double,0,1> column_vector;

// ----------------------------------------------------------------------------------------

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
#define STD_DEV 1.0

unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
default_random_engine generator (seed);


float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
    float* fp = new float[6]; 
    memcpy(fp,ppm,6*sizeof(float)); 
    for(int i = 0; i < 6; i++)
    {
        fp[i] *= (FIELD_STRENGTH*42.577);
    }
    return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};

double signal (const column_vector& acq_params, const column_vector& unknowns)
{
    const double beta = unknowns(0);
    const double T1__F = unknowns(1);
    const double T1__W = unknowns(2);
    const double rho__F = unknowns(3);
    const double rho__W = unknowns(4);
    const double R2s = unknowns(5);
    const double phi = unknowns(6);
    const double psi = unknowns(7);

    const double TR = acq_params(0);
    const double alpha = acq_params(1);
    const double TE = acq_params(2);
    const double real_part = acq_params(3);

    // compute signal model function and return the result
    double sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    double sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    
    return (real_part)*sR+(1-real_part)*sI;
}

double residual (const std::pair<column_vector, double>& data, const column_vector& params)
{
    return signal(data.first, params) - data.second;
}

std::vector<pair<input_vector,double>> signal_pairs(const int NACQS, const int NTES, const column_vector& trs, const column_vector& tips, const column_vector& tes, const column_vector& unknowns)
{
    std::vector<pair<input_vector,double>> data_samples;
    input_vector input;
    for (int nacq = 0; nacq < NACQS; nacq++)
    {
        for (int nte = 0; nte < NTES; nte++)
        {
            for (int real_part = 1; real_part >=0; real_part--)
            {
                input = trs(nacq), tips(nacq), tes(nte), real_part;
                const double sim_sig = signal(input,unknowns);//+distribution(generator); 
                data_samples.push_back(make_pair(input,sim_sig));
            }
        }
    }
    return data_samples;
}

std::vector<pair<input_vector,double>> add_noise2signal_pairs(const std::vector<pair<input_vector,double>> data_samples, const double stddev)
{
    normal_distribution<double> distribution(0.0,stddev);
    std::vector<pair<input_vector,double>> noisy_data_samples;
    for (auto ind_pair : data_samples)
    {
        const input_vector input = ind_pair.first;
        double sim_sig = ind_pair.second;
        noisy_data_samples.push_back(make_pair(input,sim_sig+distribution(generator)));
    }
    return noisy_data_samples;
}


column_vector NLSQ(std::vector<pair<input_vector,double>> noiseless_data_samples, const double std_dev_noise, const parameter_vector& unknowns)
{
    std::vector<pair<input_vector,double>> noisy_data_samples = add_noise2signal_pairs(noiseless_data_samples,std_dev_noise);
    parameter_vector x;
    x=1, 300e-3, 800e-3, 5000, 5000, 0, 0, 0;
    // cout << "Use Levenberg-Marquardt, approximate derivatives" << endl;
    solve_least_squares_lm(objective_delta_stop_strategy(1e-10)/*.be_verbose()*/, 
                           residual,
                           derivative(residual),
                           noisy_data_samples,
                           x);
    return x;
    // cout << "inferred parameters: "<< trans(x) << endl;
    // cout << "solution error:      "<< length(x - unknowns) << endl;
    // cout << endl;
}

int main(int argc, char const *argv[])
{
    int NUMTHREADS;
    int NSIMS;
    int NUMTRIALS;

    if (argc<3)
    {
        cout<<"Usage ./main NUMTHREADS NSIMS NUMTRIALS\n";
        return 1;
    }
    else
    {
        NUMTHREADS = atoi(argv[1]);
        NSIMS = atoi(argv[2]);
        NUMTRIALS = atoi(argv[3]);

    }

    int NACQS = 3; // Number of TR/FlipAngle Pairs
    int NTES = 6; // Number of Echoes

    column_vector trs(NACQS); trs = 5e-3,10e-3,15e-3;
    column_vector tips(NACQS); tips = 6*PI/180,12*PI/180,80*PI/180;
    column_vector tes(NTES); tes = 1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3;

    double beta=1.0, T1__F=312e-3, T1__W=822e-3, rho__F=50, rho__W=9050, R2s=30, phi=0, psi=0;
    parameter_vector unknowns; unknowns = beta, T1__F, T1__W, rho__F, rho__W, R2s, phi, psi;

    std::vector<pair<input_vector,double>> noiseless_data_samples = signal_pairs(NACQS, NTES, trs, tips, tes, unknowns);
    
    struct stopwatch sw;

    double betas[NSIMS]{};

    std::vector<double> timings;
    for (int trial = 0; trial < NUMTRIALS; trial++)
    {
        sw.click();
        #pragma omp parallel num_threads(NUMTHREADS)
        {
            #pragma omp for nowait
            for (int i = 0; i < NSIMS; i++)
            {
                column_vector x = NLSQ(noiseless_data_samples, 1.0, unknowns);
                betas[i] = x(0);
            }
        }
        sw.click();
        timings.push_back(sw.check());
    }
    column_vector t = mat(timings);
    double minTIME = dlib::min(t);
    cout<<NSIMS<<", "<<NUMTHREADS<<", "<<minTIME/1000.0<<"s"<<", <"<<NUMTRIALS<<" trial(s)>"<<endl;

    return 0;
}

#include "hip/hip_runtime.h"
// The contents of this file are in the public domain. See LICENSE_FOR_EXAMPLE_PROGRAMS.txt
/*

    This is an example illustrating the use the general purpose non-linear 
    least squares optimization routines from the dlib C++ Library.

    This example program will demonstrate how these routines can be used for data fitting.
    In particular, we will generate a set of data and then use the least squares  
    routines to infer the parameters of the model which generated the data.
*/


#include "./dlib-19.7/dlib/optimization.h"
#include <iostream>
#include <vector>


using namespace std;
using namespace dlib;

// ----------------------------------------------------------------------------------------

typedef matrix<double,2,1> input_vector;
typedef matrix<double,3,1> parameter_vector;
typedef matrix<double,0,1> column_vector;

// ----------------------------------------------------------------------------------------

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
    float* fp = new float[6]; 
    memcpy(fp,ppm,6*sizeof(float)); 
    for(int i = 0; i < 6; i++)
    {
        fp[i] *= (FIELD_STRENGTH*42.577);
    }
    return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};

column_vector signal (const column_vector& unknowns, const column_vector& trs, const column_vector& tips, const column_vector& tes)
{


    const double beta = unknowns(0);
    const double T1__F = unknowns(1);
    const double T1__W = unknowns(2);
    const double rho__F = unknowns(3);
    const double rho__W = unknowns(4);
    const double R2s = unknowns(5);
    const double phi = unknowns(6);
    const double psi = unknowns(7);

    double TR;
    double alpha;
    double TE;
    double sR;
    double sI;

    std::vector<double> v;
    // compute signal model function and return the result
    for (int ACQNUM = 0; ACQNUM < trs.nr(); ACQNUM++)
    {
        for (int TENUM = 0; TENUM < tes.nr(); TENUM++)
        {
            TR = trs(ACQNUM);
            alpha = tips(ACQNUM);
            TE = tes(TENUM);
            sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
            sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
            v.push_back(sR);
            v.push_back(sI);
        }
    }
    
    matrix<double,0,1> sig = mat(v);
    return sig;
}

// We will use this function to generate data.  It represents a function of 2 variables
// and 3 parameters.   The least squares procedure will be used to infer the values of 
// the 3 parameters based on a set of input/output pairs.
double model (
    const input_vector& input,
    const parameter_vector& params
)
{
    const double p0 = params(0);
    const double p1 = params(1);
    const double p2 = params(2);

    const double i0 = input(0);
    const double i1 = input(1);

    const double temp = p0*i0 + p1*i1 + p2;

    return temp*temp;
}

// ----------------------------------------------------------------------------------------

// This function is the "residual" for a least squares problem.   It takes an input/output
// pair and compares it to the output of our model and returns the amount of error.  The idea
// is to find the set of parameters which makes the residual small on all the data pairs.
double residual (
    const std::pair<input_vector, double>& data,
    const parameter_vector& params
)
{
    return model(data.first, params) - data.second;
}

double my_residual (
    const std::pair<input_vector, double>& data,
    const parameter_vector& params
)
{
    return model(data.first, params) - data.second;
}


// ----------------------------------------------------------------------------------------

// This function is the derivative of the residual() function with respect to the parameters.
parameter_vector residual_derivative (
    const std::pair<input_vector, double>& data,
    const parameter_vector& params
)
{
    parameter_vector der;

    const double p0 = params(0);
    const double p1 = params(1);
    const double p2 = params(2);

    const double i0 = data.first(0);
    const double i1 = data.first(1);

    const double temp = p0*i0 + p1*i1 + p2;

    der(0) = i0*2*temp;
    der(1) = i1*2*temp;
    der(2) = 2*temp;

    return der;
}

// ----------------------------------------------------------------------------------------

int main()
{
    // column_vector acq_params(3);
    // acq_params = 20e-3, 20*PI/180, 1.2e-3;

    int NPs = 3; // Number of TR/FlipAngle Pairs
    int NTEs = 6; // Number of Echoes
    int NACQS = NPs*NTEs;

    column_vector tes(NTEs); tes = 1.2e-3,3.2e-3,5.2e-3,7.2e-3,9.2e-3,11.2e-3;
    column_vector trs(NPs); trs = 5e-3,10e-3,15e-3;
    column_vector tips(NPs); tips = 6*PI/180,12*PI/180,80*PI/180;

    column_vector unknowns(8);
    unknowns = 1,312e-3,822e-3,50,950,30,0,0;

    cout<<signal(unknowns, trs, tips, tes);

    try
    {
        // randomly pick a set of parameters to use in this example
        const parameter_vector params = 10*randm(3,1);
        cout << "params: " << trans(params) << endl;


        // Now let's generate a bunch of input/output pairs according to our model.
        std::vector<std::pair<input_vector, double> > data_samples;
        input_vector input;
        for (int i = 0; i < 1000; ++i)
        {
            input = 10*randm(2,1);
            const double output = model(input, params);

            // save the pair
            data_samples.push_back(make_pair(input, output));
        }

        // Before we do anything, let's make sure that our derivative function defined above matches
        // the approximate derivative computed using central differences (via derivative()).  
        // If this value is big then it means we probably typed the derivative function incorrectly.
        cout << "derivative error: " << length(residual_derivative(data_samples[0], params) - 
                                               derivative(residual)(data_samples[0], params) ) << endl;





        // Now let's use the solve_least_squares_lm() routine to figure out what the
        // parameters are based on just the data_samples.
        parameter_vector x;
        x = 1;

        cout << "Use Levenberg-Marquardt" << endl;
        // Use the Levenberg-Marquardt method to determine the parameters which
        // minimize the sum of all squared residuals.
        solve_least_squares_lm(objective_delta_stop_strategy(1e-7).be_verbose(), 
                               residual,
                               residual_derivative,
                               data_samples,
                               x);

        // Now x contains the solution.  If everything worked it will be equal to params.
        cout << "inferred parameters: "<< trans(x) << endl;
        cout << "solution error:      "<< length(x - params) << endl;
        cout << endl;




        x = 1;
        cout << "Use Levenberg-Marquardt, approximate derivatives" << endl;
        // If we didn't create the residual_derivative function then we could
        // have used this method which numerically approximates the derivatives for you.
        solve_least_squares_lm(objective_delta_stop_strategy(1e-7).be_verbose(), 
                               residual,
                               derivative(residual),
                               data_samples,
                               x);

        // Now x contains the solution.  If everything worked it will be equal to params.
        cout << "inferred parameters: "<< trans(x) << endl;
        cout << "solution error:      "<< length(x - params) << endl;
        cout << endl;




        x = 1;
        cout << "Use Levenberg-Marquardt/quasi-newton hybrid" << endl;
        // This version of the solver uses a method which is appropriate for problems
        // where the residuals don't go to zero at the solution.  So in these cases
        // it may provide a better answer.
        solve_least_squares(objective_delta_stop_strategy(1e-7).be_verbose(), 
                            residual,
                            residual_derivative,
                            data_samples,
                            x);

        // Now x contains the solution.  If everything worked it will be equal to params.
        cout << "inferred parameters: "<< trans(x) << endl;
        cout << "solution error:      "<< length(x - params) << endl;

    }
    catch (std::exception& e)
    {
        cout << e.what() << endl;
    }
}

// Example output:
/*
params: 8.40188 3.94383 7.83099 

derivative error: 9.78267e-06
Use Levenberg-Marquardt
iteration: 0   objective: 2.14455e+10
iteration: 1   objective: 1.96248e+10
iteration: 2   objective: 1.39172e+10
iteration: 3   objective: 1.57036e+09
iteration: 4   objective: 2.66917e+07
iteration: 5   objective: 4741.9
iteration: 6   objective: 0.000238674
iteration: 7   objective: 7.8815e-19
iteration: 8   objective: 0
inferred parameters: 8.40188 3.94383 7.83099 

solution error:      0

Use Levenberg-Marquardt, approximate derivatives
iteration: 0   objective: 2.14455e+10
iteration: 1   objective: 1.96248e+10
iteration: 2   objective: 1.39172e+10
iteration: 3   objective: 1.57036e+09
iteration: 4   objective: 2.66917e+07
iteration: 5   objective: 4741.87
iteration: 6   objective: 0.000238701
iteration: 7   objective: 1.0571e-18
iteration: 8   objective: 4.12469e-22
inferred parameters: 8.40188 3.94383 7.83099 

solution error:      5.34754e-15

Use Levenberg-Marquardt/quasi-newton hybrid
iteration: 0   objective: 2.14455e+10
iteration: 1   objective: 1.96248e+10
iteration: 2   objective: 1.3917e+10
iteration: 3   objective: 1.5572e+09
iteration: 4   objective: 2.74139e+07
iteration: 5   objective: 5135.98
iteration: 6   objective: 0.000285539
iteration: 7   objective: 1.15441e-18
iteration: 8   objective: 3.38834e-23
inferred parameters: 8.40188 3.94383 7.83099 

solution error:      1.77636e-15
*/

#include "hip/hip_runtime.h"
// The contents of this file are in the public domain. See LICENSE_FOR_EXAMPLE_PROGRAMS.txt
/*

    This is an example illustrating the use the general purpose non-linear 
    optimization routines from the dlib C++ Library.

    The library provides implementations of the conjugate gradient,  BFGS,
    L-BFGS, and BOBYQA optimization algorithms.  These algorithms allow you to
    find the minimum of a function of many input variables.  This example walks
    though a few of the ways you might put these routines to use.

*/


#include "./dlib-19.7/dlib/optimization.h"
#include <iostream>


using namespace std;
using namespace dlib;

// ----------------------------------------------------------------------------------------

// In dlib, the general purpose solvers optimize functions that take a column
// vector as input and return a double.  So here we make a typedef for a
// variable length column vector of doubles.  This is the type we will use to
// represent the input to our objective functions which we will be minimizing.
typedef matrix<double,0,1> column_vector;

// ----------------------------------------------------------------------------------------
// Below we create a few functions.  When you get down into main() you will see that
// we can use the optimization algorithms to find the minimums of these functions.
// ----------------------------------------------------------------------------------------

#define FIELD_STRENGTH 3.0
#define PI 3.141592654
float ppm[6] = {0.6, -0.5, -1.95, -2.60, -3.40, -3.80};

float* fp_maker(const float* ppm)
{
    float* fp = new float[6]; 
    memcpy(fp,ppm,6*sizeof(float)); 
    for(int i = 0; i < 6; i++)
    {
        fp[i] *= (FIELD_STRENGTH*42.577);
    }
    return fp;
}

float* fp = fp_maker(ppm);
float a[6] = {4.7/100, 3.9/ 100, 0.6/ 100, 12.0/ 100, 70.0/ 100, 8.8/ 100};



float* signal_equation(const float TR, const float alpha, const float TE, const float beta, const float T1__F, const float T1__W, const float rho__F, const float rho__W, const float R2s, const float phi, const float psi)
{   
    float sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    float sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    
    float* sig = new float[2];
    sig[0] = sR;
    sig[1] = sI;
    return sig;
}

float* signal (const column_vector& params, const column_vector& acq)
{
    const double TR = acq(0);
    const double alpha = acq(1);
    const double TE = acq(2);

    const double beta = params(0);
    const double T1__F = params(1);
    const double T1__W = params(2);
    const double rho__F = params(3);
    const double rho__W = params(4);
    const double R2s = params(5);
    const double phi = params(6);
    const double psi = params(7);


    // compute signal model function and return the result
    float sR = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * cos(TE * psi + phi) - rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * sin(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    float sI = ((rho__W * sin(beta * alpha) * (1.0 - exp(-TR / T1__W)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__W)) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * cos(2.0 * PI * fp[0] * TE) + a[1] * cos(2.0 * PI * fp[1] * TE) + a[2] * cos(2.0 * PI * fp[2] * TE) + a[3] * cos(2.0 * PI * fp[3] * TE) + a[4] * cos(2.0 * PI * fp[4] * TE) + a[5] * cos(2.0 * PI * fp[5] * TE)) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * sin(TE * psi + phi) + rho__F * sin(beta * alpha) * (1.0 - exp(-TR / T1__F)) * (a[0] * sin(2.0 * PI * fp[0] * TE) + a[1] * sin(2.0 * PI * fp[1] * TE) + a[2] * sin(2.0 * PI * fp[2] * TE) + a[3] * sin(2.0 * PI * fp[3] * TE) + a[4] * sin(2.0 * PI * fp[4] * TE) + a[5] * sin(2.0 * PI * fp[5] * TE)) * cos(TE * psi + phi) / (1.0 - cos(beta * alpha) * exp(-TR / T1__F))) * exp(-R2s * TE);
    
    float* sig = new float[2];
    sig[0] = sR;
    sig[1] = sI;
    return sig;
}

// ----------------------------------------------------------------------------------------

class test_function
{
public:

    test_function (const column_vector& input)
    {
        target = input;
    }

    double operator() ( const column_vector& arg) const
    {
        // return the mean squared error between the target vector and the input vector
        return mean(squared(target-arg));
    }

private:
    column_vector target;
};

// ----------------------------------------------------------------------------------------


int main(int argc, char const *argv[])
{
    /* code */
    column_vector target(4);
    target = 1,2,3,4;
    struct test_function tf = test_function(target);
    target = 2,3,4,5;
    cout<<tf(target)<<endl;

    return 0;
}

